#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "./p3.h"

__device__ uint get_smid(void) {
  uint ret;
  asm("mov.u32 %0, %smid;" : "=r"(ret) );
  return ret;
}

//Sid: Function to be called on device for addition of the elements from queue[3] to queue[103] and so on
__device__ void* calcValue(void *voidQueue) {
   if (threadIdx.x <101) {
      int sm = get_smid();
      taskQueue_t *queues = (taskQueue_t*)voidQueue;

      int size = queues[(sm*104)+1].val;
      int thread_no = queues[624].val;

      int steps = size/thread_no;
      int rem = size-(steps*thread_no);
      int i = 0;
      int start_index = (sm*104)+3;

      if (threadIdx.x < rem) {
         steps+=1;
      }
      if (threadIdx.x < size) {
         for (i = 0; i<steps; i++) {
             atomicAdd(&queues[(sm*104)+2].val, queues[start_index+(i*thread_no)+threadIdx.x].val);
             queues[start_index+(i*thread_no)+threadIdx.x].taskDone = 1;
         }
      }
      __syncthreads();
      //Sid:Check that all tasks for the SM are done
      for( i = start_index; i<=(start_index+100); i++){
         if (queues[i].taskDone = 0) {
            queues[sm*104].taskDone = 0;
            break;
         } else {
            queues[sm*104].taskDone = 1;
         }
      }
   }
}

//Sid: Schedule the tasks based on the task_done flag
__global__ void scheduler(taskQueue_t *queues) {
 //Sid: Limiting this to 101 as the array size is limited to 100 data only
 if (threadIdx.x <101) {
   /*Sid: Call the function based on the func_no: example func_no = 0 is addition of tasks/data.
    *     This can be extended to include other functionalities as well.
    */
   int sm = get_smid();
   if (queues[sm*104].func == 0 && queues[sm*104].taskDone == 0) {
      calcValue(queues);
   }
 }
}

//Sid:Add tasks to the individual SM's queues initially
int taskAdd(void *(*func) (void *), void *arg, int sm) {
  int index = sm*104;
  int task_size = ((int*)arg)[0];
  int i =0;
  //Sid: SM no.
  queue[index].val = sm;
  //Sid: func = 0 is for addition.This implementation is limited to Addition only. We can extend it to other functionalities as well
  queue[index].func = 0;
  queue[index].taskDone = 0;
  gettimeofday(&queue[index].start,NULL);
  gettimeofday(&queue[index++].end,NULL);
  //Sid: Size of Task i.e. No of Tasks to be performed
  if (task_size == 200){
     if (sm == 0 || sm == 1){
        task_size = 100;
     }else if (sm == 2 || sm == 3){
        task_size = 50;
     }else {
        task_size = 10;
     }
  }
  queue[index++].val = task_size;
  //Sid: Initialize Sum
  queue[index++].val = 0;
  //Sid: Set the array values
  for (i=0; i < task_size; i++) {
      queue[index].taskDone = 0;
      //Sid: Tasks in this implementation are addition of data from 0 to task_size -1
      queue[index++].val = i;
  }
  //Sid: Set the task done flag for all to 0
  queue[index++].val = 0;
  return SUCCESS;
}

//Sid: Check Task Done
int taskDone(int taskId) {
    return queue[taskId].taskDone;
}
//Sid:Below Code for calculating time interval Taken from Homework3 TfIDf program
long calcDiffTime(struct timeval* strtTime, struct timeval* endTime) {

return( endTime->tv_sec*1000000 + endTime->tv_usec - strtTime->tv_sec*1000000 - strtTime->tv_usec );
}

extern "C" int call_sched(int M, int N, int task) {

  int args[2];
  int wait = 0;
  args[0]=task;
  args[1]=N;
  //Sid: Device Queue
  taskQueue_t *dev_queue;
  int i = 0;
  long DiffTime;
  int count = 0;
  //Sid: Initialize task queue
  for (i = 0; i<MAX_TASK; i++) {
     queue[i].val = 0;
  }
  
  int size = MAX_TASK*(sizeof(taskQueue_t));
  hipMalloc((void**)&dev_queue, size);
  //Sid: Add tasks to the queues
  for (i = 0; i<M; i++) {
     taskAdd(calcValue,&args, i);
  }
  
  queue[624].val = N;
  hipMemcpyAsync(dev_queue, queue, size, hipMemcpyHostToDevice,0);
  scheduler<<<M,N>>>( dev_queue);

  //Sid: Wait here and check that all tasks are done
  while (wait == 0) {
  //Sid: Update the local queue.
  hipMemcpyAsync(queue, dev_queue, size, hipMemcpyDeviceToHost,0);
     for (i = 0; i <M; i++) {
       if (taskDone(i*104) == 1) {
          gettimeofday(&queue[i*104].end, NULL);
          count+=1;
          DiffTime = calcDiffTime(&(queue[i*104].start), &(queue[i*104].end));
          printf("SM No. %d Completed!\nTime Taken From the time task is added to SM finishes task: %ld\nxxxxxxxxxxxx\n",i,DiffTime);
       }
     }
     if (count == 6) {
        wait = 1;
     } else {
        count = 0;
     }
  }
  hipDeviceSynchronize();
  //Sid:Copy back results to host when all SM are finished
  hipMemcpyAsync(queue, dev_queue, size, hipMemcpyDeviceToHost,0);
  
  //Sid: Right now all SM are doing the same thing so we can just check one SMs function and print output accordingly for all SMs
  if (queue[0].func == 0) {
     if (task == 200) {
        printf("Task:Addition of Array of size 100 in SM 0&1. 50 in SM 2&3. 10 in SM 4&5! \nValues from 0 to 99|0 to 49|0 to 9\nSM: Output:\n");
     }else{
        printf("Task:Addition of Array of size %d! \nValues from 0 to %d\nSM: Output:\n",task,(task-1));
     }

     for (i=0;i<M;i++){
        printf("%d   %d \n",i,queue[(i*104)+2].val);
     }
  }
  //Sid: Cleanup
  hipFree(dev_queue);
  return SUCCESS;
}
